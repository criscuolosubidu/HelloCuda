#include <iostream>
#include <hip/hip_runtime.h>
#include <numeric>
#include <random>
#include <vector>
#include <chrono>

#define WARP_SIZE 32
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])

template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val) {
#pragma unroll
    for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
        val += __shfl_xor_sync(0xffffffff, val, mask);
    }
    return val;
}

template<const int NUM_THREADS = 256>
__global__ void block_all_reduce_sum_f32_f32_kernel(float *a, float *y, int N) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * NUM_THREADS + threadIdx.x;
    constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
    __shared__ float reduce_sum[NUM_WARPS];
    float sum = (idx < N) ? a[idx] : 0.0f;
    int warp = tid / WARP_SIZE;
    int lane = tid % WARP_SIZE;
    sum = warp_reduce_sum_f32<WARP_SIZE>(sum);
    if (lane == 0) reduce_sum[warp] = sum;
    __syncthreads();
    sum = (lane < NUM_WARPS) ? reduce_sum[lane] : 0.0f;
    if (warp == 0) sum = warp_reduce_sum_f32<NUM_WARPS>(sum);
    if (tid == 0) atomicAdd(y, sum);
}

// a: N x 1, y = sum(a)
template<const int NUM_THREADS = 256 / 4>
__global__ void block_all_reduce_sum_f32x4_f32_kernel(float *a, float *y, int N) {
    int tid = threadIdx.x;
    int idx = (blockIdx.x * NUM_THREADS + threadIdx.x) * 4;
    constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
    __shared__ float reduce_sum[NUM_WARPS];
    // read as float4 type
    float4 reg_a = FLOAT4(a[idx]);
    float sum = (idx < N) ? reg_a.x + reg_a.y + reg_a.z + reg_a.w : 0.0f;
    int warp = tid / WARP_SIZE;
    int lane = tid % WARP_SIZE;
    sum = warp_reduce_sum_f32<WARP_SIZE>(sum);
    if (lane == 0) reduce_sum[warp] = sum;
    __syncthreads();
    sum = (lane < NUM_WARPS) ? reduce_sum[lane] : 0.0f;
    if (warp == 0) sum = warp_reduce_sum_f32<NUM_WARPS>(sum);
    if (tid == 0) atomicAdd(y, sum);
}


int main() {
    std::cout << "test cuda kernels!" << std::endl;
    constexpr int N = 256 * 256;

    // CPU part
    std::vector<float> h_x(N);
    float h_y = 0.0f;
    std::mt19937 gen(std::random_device{}());
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);
    for (int i = 0; i < N; ++i) {
        h_x[i] = dist(gen);
    }

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, sizeof(float));

    auto start = std::chrono::high_resolution_clock::now();

    hipMemcpy(d_x, h_x.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_y, 0, sizeof(float));

    // block_all_reduce_sum_f32x4_f32_kernel<<<256, 256/4>>>(d_x, d_y, N); // 4.3913 ms
    block_all_reduce_sum_f32_f32_kernel<<<256, 256>>>(d_x, d_y, N); // 8.3018 ms

    hipDeviceSynchronize();

    hipMemcpy(&h_y, d_y, sizeof(float), hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> elapsed_ms = end - start;
    std::cout << "GPU operations took: " << elapsed_ms.count() << " ms\n";

    float ans = accumulate(h_x.begin(), h_x.end(), 0.0f);
    std::cout << "CPU sum: " << ans << ", GPU sum: " << h_y << std::endl;

    hipFree(d_x);
    hipFree(d_y);

    return 0;
}