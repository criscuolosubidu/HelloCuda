#include <iostream>
#include <hip/hip_runtime.h>
#include <numeric>
#include <random>
#include <vector>

#define WARP_SIZE 32

template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float val) {
#pragma unroll
    for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1) {
        val += __shfl_xor_sync(0xffffffff, val, mask);
    }
    return val;
}

template<const int NUM_THREADS = 256>
__global__ void block_all_reduce_sum_f32_f32_kernel(float *a, float *y, int N) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * NUM_THREADS + threadIdx.x;
    constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
    __shared__ float reduce_sum[NUM_WARPS];
    float sum = (idx < N) ? a[idx] : 0.0f;
    int warp = tid / WARP_SIZE;
    int lane = tid % WARP_SIZE;
    sum = warp_reduce_sum_f32<WARP_SIZE>(sum);
    if (lane == 0) reduce_sum[warp] = sum;
    __syncthreads();
    sum = (lane < NUM_WARPS) ? reduce_sum[lane] : 0.0f;
    if (warp == 0) sum = warp_reduce_sum_f32<NUM_WARPS>(sum);
    if (tid == 0) atomicAdd(y, sum);
}


int main() {
    std::cout << "test cuda kernels!" << std::endl;
    constexpr int N = 256 * 256;

    // CPU part
    std::vector<float> h_x(N);
    float h_y = 0.0f;
    std::mt19937 gen(std::random_device{}());
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);
    for (int i = 0; i < N; ++i) {
        h_x[i] = dist(gen);
    }

    float *d_x, *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, sizeof(float));

    hipMemcpy(d_x, h_x.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_y, 0, sizeof(float));

    block_all_reduce_sum_f32_f32_kernel<<<256, 256>>>(d_x, d_y, N);
    hipMemcpy(&h_y, d_y, sizeof(float), hipMemcpyDeviceToHost);

    float ans = accumulate(h_x.begin(), h_x.end(), 0.0f);
    std::cout << ans << ' ' << h_y << std::endl;

    hipFree(d_x);
    hipFree(d_y);
}

