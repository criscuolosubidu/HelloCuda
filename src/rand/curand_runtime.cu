#include <iostream>
#include <hip/hip_runtime.h>
#include <numeric>
#include <random>
#include <vector>
#include <chrono>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <thread>
#include <hiprand/hiprand.h>

// Scale uniform [0,1) to [min, max)
__global__ void scale_uniform_kernel(float *data, int N, float min_val, float max_val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data[idx] = data[idx] * (max_val - min_val) + min_val;
    }
}

int main() {
    constexpr int N = 1073741824; // 2 ^ 30
    constexpr int NUM_THREADS = 256;
    constexpr int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

    float h_y;
    std::vector<float> h_x(N);

    auto start_gen = std::chrono::high_resolution_clock::now();

    float *d_data, *h_data;

    hipMalloc(&d_data, N * sizeof(float));
    h_data = new float[N];

    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1233ULL);
    hiprandGenerateUniform(gen, d_data, N);

    // choose to scale
    constexpr float MIN_VAL = 0.0f;
    constexpr float MAX_VAL = 0.01f;
    int num_blocks = (N + 255) / 256;
    scale_uniform_kernel<<<num_blocks, 256>>>(d_data, N, MIN_VAL, MAX_VAL);

    hipMemcpy(h_data, d_data, N * sizeof(float), hipMemcpyDeviceToHost);

    hiprandDestroyGenerator(gen);
    hipFree(d_data);
    delete[] h_data;

    auto end_gen = std::chrono::high_resolution_clock::now();
    auto gen_time = std::chrono::duration<double, std::milli>(end_gen - start_gen).count();

    std::cout << "GPU Random Number Generate Time : " << gen_time << " ms" << std::endl; // 943 ms
}