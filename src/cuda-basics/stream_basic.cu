#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>
#include <string_view>
#include <sstream>
#include <vector>
#include <random>

class CudaError : public std::runtime_error {
public:
    CudaError(hipError_t code, std::string_view func, std::string_view file, int line)
        : std::runtime_error(format(code, func, file, line)), code_(code) {}

    [[nodiscard]] hipError_t code() const { return code_; }

private:
    hipError_t code_;
    static std::string format(hipError_t code, std::string_view func, std::string_view file, int line) {
        std::ostringstream oss;
        oss << "CUDA error [" << hipGetErrorName(code) << "]: "
            << hipGetErrorString(code) << "\n  in " << func
            << "\n  at " << file << ":" << line;
        return oss.str();
    }
};

inline void checkCuda(hipError_t err, std::string_view func, std::string_view file, int line) {
    if (err != hipSuccess) {
        throw CudaError(err, func, file, line);
    }
}

__global__ void vectorAdd(const float* A, const float* B, float* C, unsigned int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) C[i] = A[i] + B[i];
}

int streamBasicsDemo() {
    try {
        constexpr unsigned int n = (1u << 20);
        constexpr size_t sz = n * sizeof(float);

        std::vector<float> h_A(n), h_B(n), h_C(n);

        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_real_distribution<float> dis(0.0f, 1.0f);
        for (auto& v : h_A) v = dis(gen);
        for (auto& v : h_B) v = dis(gen);

        float *d_A, *d_B, *d_C;
        checkCuda(hipMalloc((void**)&d_A, sz), "hipMalloc", __FILE__, __LINE__);
        checkCuda(hipMalloc((void**)&d_B, sz), "hipMalloc", __FILE__, __LINE__);
        checkCuda(hipMalloc((void**)&d_C, sz), "hipMalloc", __FILE__, __LINE__);

        hipStream_t stream1, stream2;
        checkCuda(hipStreamCreate(&stream1), "hipStreamCreate", __FILE__, __LINE__);
        checkCuda(hipStreamCreate(&stream2), "hipStreamCreate", __FILE__, __LINE__);

        checkCuda(hipMemcpyAsync(d_A, h_A.data(), sz, hipMemcpyHostToDevice, stream1), "hipMemcpyAsync", __FILE__, __LINE__);
        checkCuda(hipMemcpyAsync(d_B, h_B.data(), sz, hipMemcpyHostToDevice, stream2), "hipMemcpyAsync", __FILE__, __LINE__);

        constexpr unsigned int threadsPerBlock = 256;
        constexpr unsigned int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

        vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(d_A, d_B, d_C, n);
        checkCuda(hipMemcpyAsync(h_C.data(), d_C, sz, hipMemcpyDeviceToHost, stream1), "hipMemcpyAsync", __FILE__, __LINE__);

        checkCuda(hipStreamSynchronize(stream1), "hipStreamSynchronize", __FILE__, __LINE__);
        checkCuda(hipStreamSynchronize(stream2), "hipStreamSynchronize", __FILE__, __LINE__);

        for (size_t i = 0; i < n; ++i) {
            if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
                throw std::runtime_error("Verification failed at index" + std::to_string(i));
            }
        }

        std::cout << "Test Passed!" << std::endl;

    } catch (const CudaError& e) {
        std::cerr << "CUDA Error:\n" << e.what() << std::endl;
        return EXIT_FAILURE;
    } catch (const std::exception& e) {
        std::cerr << "Exception: " << e.what() << std::endl;
        return EXIT_FAILURE;
    }
    return EXIT_SUCCESS;
}



