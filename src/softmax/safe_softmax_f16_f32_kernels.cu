#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <thread>
#include <hiprand.h>

#define WARP_SIZE 32
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])


// Scale uniform [0,1) to [min, max)
__global__ void scale_uniform_kernel(float *data, int N, float min_val, float max_val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data[idx] = data[idx] * (max_val - min_val) + min_val;
    }
}

// transform f32 to f16
__global__ void transform_f32_f16_kernel(float *data, half *data_f16, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data_f16[idx] = __float2half(data[idx]);
    }
}

struct __align__(8) MD {
    float m;
    float d;
};

// Warp Reduce for Online Softmax
template <const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ MD warp_reduce_md_op(MD value) {
    unsigned int mask = 0xffffffff;
#pragma unroll
    for (int stride = kWarpSize >> 1; stride > 0; stride >>= 1) {
        MD other{};
        other.m = __shfl_xor_sync(mask, value.m, stride);
        other.d = __shfl_xor_sync(mask, value.d, stride);
        bool is_big = value.m > other.m;
        MD bigger_md = is_big ? value : other;
        MD smaller_md = is_big ? other : value;
        value.m = bigger_md.m;
        value.d = bigger_md.d + smaller_md.d * __expf(smaller_md.m - bigger_md.m);
    }
    return value;
}

// Warp Reduce for sum
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum_f32(float value) {
#pragma unroll
    for (int stride = kWarpSize >> 1; stride > 0; stride >>= 1) {
        value += __shfl_xor_sync(0xffffffff, value, stride);
    }
    return value;
}

// Warp Reduce for max
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_max_f32(float value) {
#pragma unroll
    for (int stride = kWarpSize >> 1; stride > 0; stride >>= 1) {
        value = fmaxf(value, __shfl_xor_sync(0xffffffff, value, stride));
    }
    return value;
}

// grid 1D
// grid(N / 256), block(256)
template<const int NUM_THREADS = 256>
__device__ float block_reduce_sum_f32(float val) {
    constexpr int NUM_WAPRS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
    static __shared__ float reduce_sum[NUM_WAPRS];
    int tid = threadIdx.x;
    int warp = tid / WARP_SIZE;
    int lane = tid % WARP_SIZE;
    float value = warp_reduce_sum_f32<WARP_SIZE>(val);
    if (lane == 0) reduce_sum[warp] = value;
    __syncthreads();
    value = lane < NUM_WAPRS ? reduce_sum[lane] : 0.0f;
    value = warp_reduce_sum_f32<NUM_WAPRS>(value);
    // if you choose `value = warp_reduce_sum_f32<WARP_SIZE>(value)` then you don't need to sync them explicitly
    value = __shfl_sync(0xffffffff, value, 0, 32);
    return value;
}

// grid 1D
// grid(N / 256), block(256)
template<const int NUM_THREADS = 256>
__device__ float block_reduce_max_f32(float val) {
    constexpr int NUM_WAPRS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
    static __shared__ float reduce_max[NUM_WAPRS];
    int tid = threadIdx.x;
    int warp = tid / WARP_SIZE;
    int lane = tid % WARP_SIZE;
    float value = warp_reduce_max_f32<WARP_SIZE>(val);
    if (lane == 0) reduce_max[warp] = value;
    __syncthreads();
    value = lane < NUM_WAPRS ? reduce_max[lane] : -FLT_MAX;
    value = warp_reduce_max_f32<NUM_WAPRS>(value);
    // if you choose `value = warp_reduce_sum_f32<WARP_SIZE>(value)` then you don't need to sync them explicitly
    value = __shfl_sync(0xffffffff, value, 0, 32);
    return value;
}


template<const int NUM_THREADS = 256>
__global__ void safe_softmax_f16_f32_per_token_kernel(half *x, half *y, int N) {
    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float mx_val = idx < N ? __half2float(x[idx]) : -FLT_MAX;
    mx_val = block_reduce_max_f32<NUM_THREADS>(mx_val);
    float exp_val = idx < N ? expf(__half2float(x[idx]) - mx_val) : 0.0f;
    float exp_sum = block_reduce_sum_f32<NUM_THREADS>(exp_val);
    if (idx < N) y[idx] = __float2half(exp_val / exp_sum);
}


template<const int NUM_THREADS = 256 / 2>
__global__ void safe_softmax_f16x2_f32_per_token_kernel(half *x, half *y, int N) {
    const int tid = threadIdx.x;
    const int idx = (blockIdx.x * blockDim.x + tid) * 2;

    // read 2 half, turn into 2 float
    float2 reg_x = __half22float2(HALF2(x[idx]));
    float max_val = -FLT_MAX;
    max_val = idx + 0 < N ? fmaxf(reg_x.x, max_val) : -FLT_MAX;
    max_val = idx + 1 < N ? fmaxf(reg_x.y, max_val) : -FLT_MAX;
    max_val = block_reduce_max_f32<NUM_THREADS>(max_val); // block max

    float2 reg_exp;
    reg_exp.x = idx + 0 < N ? expf(reg_x.x - max_val) : 0.0f;
    reg_exp.y = idx + 1 < N ? expf(reg_x.y - max_val) : 0.0f;
    float exp_val = reg_exp.x + reg_exp.y;
    float exp_sum = block_reduce_sum_f32<NUM_THREADS>(exp_val); // block sum

    float2 reg_y;
    reg_y.x = reg_exp.x / exp_sum;
    reg_y.y = reg_exp.y / exp_sum;
    if (idx + 1 < N)
        HALF2(y[idx]) = __float22half2_rn(reg_y);
}


template<const int NUM_THREADS = 256 / 8>
__global__ void safe_softmax_f16x8_pack_f32_per_token_kernel(half *x, half *y, int N) {
    const int tid = threadIdx.x;
    const int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
    half pack_x[8], pack_y[8];
    LDST128BITS(pack_x[0]) = LDST128BITS(x[idx]);
    float mx_val = -FLT_MAX;
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        mx_val = fmaxf(mx_val, __half2float(pack_x[i]));
    }
    mx_val = block_reduce_max_f32<NUM_THREADS>(mx_val);
    float exp_sum = 0.0f;
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        exp_sum += expf(__half2float(pack_x[i]) - mx_val);
    }
    exp_sum = block_reduce_sum_f32<NUM_THREADS>(exp_sum);
#pragma unroll
    for (int i = 0; i < 8; ++i) {
        float exp_val = expf(__half2float(pack_x[i]) - mx_val);
        pack_y[i] = __float2half_rn(exp_val / exp_sum);
    }
    if (idx + 7 < N) LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]);
}


int main() {
    std::cout << "Start Testing Softmax kernels" << std::endl;
    constexpr int TOKENS = 2048;      // Number of tokens
    constexpr int TOKEN_DIM = 256;    // Dimension per token
    constexpr int N = TOKENS * TOKEN_DIM; // Total elements

    std::vector<float> h_y(N);
    std::vector<float> h_x(N);

    auto start_gen = std::chrono::high_resolution_clock::now();

    float *d_x;
    float *d_y;
    half *d_x_half;
    half *d_y_half;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_x_half, N * sizeof(half));
    hipMalloc(&d_y_half, N * sizeof(half));

    // gen
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1233ULL);
    hiprandGenerateUniform(gen, d_x, N);

    // scale to [0, 1]
    constexpr float MIN_VAL = 0.0f;
    constexpr float MAX_VAL = 2;
    scale_uniform_kernel<<<TOKENS, TOKEN_DIM>>>(d_x, N, MIN_VAL, MAX_VAL);

    hipDeviceSynchronize();

    // transform fp32 -> fp16
    transform_f32_f16_kernel<<<TOKENS, TOKEN_DIM>>>(d_x, d_x_half, N);

    auto end_gen = std::chrono::high_resolution_clock::now();
    auto gen_time = std::chrono::duration<double, std::milli>(end_gen - start_gen).count();
    std::cout << "GPU Random Number Generate Time : " << gen_time << " ms" << std::endl;

    // copy to host
    auto start_copy = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_x.data(), d_x, N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    auto end_copy = std::chrono::high_resolution_clock::now();
    auto copy_time = std::chrono::duration<double, std::milli>(end_copy - start_copy).count();
    std::cout << "GPU Copy data Time : " << copy_time << " ms" << std::endl;

    std::cout << "Start Compute softmax" << std::endl;
    auto start_compute = std::chrono::high_resolution_clock::now();

    // kernel functions
    // safe_softmax_f16_f32_per_token_kernel<<<TOKENS, TOKEN_DIM>>>(d_x_half, d_y_half, N); // 82.97x speed up
    // safe_softmax_f16x2_f32_per_token_kernel<<<TOKENS, TOKEN_DIM / 2>>>(d_x_half, d_y_half, N); // 78.134x speed up
    safe_softmax_f16x8_pack_f32_per_token_kernel<<<TOKENS, TOKEN_DIM / 8>>>(d_x_half, d_y_half, N); // 90.3038x speed up

    hipDeviceSynchronize();

    auto end_compute = std::chrono::high_resolution_clock::now();
    auto compute_time = std::chrono::duration<double, std::milli>(end_compute - start_compute).count();
    std::cout << "GPU Compute softmax Time : " << compute_time << " ms" << std::endl;

    // CPU reference computation
    std::cout << "Start CPU Reference Compute" << std::endl;
    auto start_cpu = std::chrono::high_resolution_clock::now();

    // Per-token safe softmax: each token independently normalized
    for (int token = 0; token < TOKENS; ++token) {
        int base_idx = token * TOKEN_DIM;

        // Step 1: Find maximum value for this token (for numerical stability)
        float max_val = h_x[base_idx];
        for (int i = 1; i < TOKEN_DIM; ++i) {
            max_val = fmaxf(max_val, h_x[base_idx + i]);
        }

        // Step 2: Calculate sum of exp(x - max_val) for this token
        float exp_sum = 0.0f;
        for (int i = 0; i < TOKEN_DIM; ++i) {
            exp_sum += expf(h_x[base_idx + i] - max_val);
        }

        // Step 3: Normalize for this token
        for (int i = 0; i < TOKEN_DIM; ++i) {
            h_y[base_idx + i] = expf(h_x[base_idx + i] - max_val) / exp_sum;
        }
    }

    auto end_cpu = std::chrono::high_resolution_clock::now();
    auto cpu_time = std::chrono::duration<double, std::milli>(end_cpu - start_cpu).count();
    std::cout << "CPU Reference Compute Time : " << cpu_time << " ms" << std::endl;

    // Copy GPU result back to host
    std::vector<float> h_y_gpu(N);
    std::vector<half> h_y_half(N);
    hipMemcpy(h_y_half.data(), d_y_half, N * sizeof(half), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for (int i = 0; i < N; ++i) h_y_gpu[i] = __half2float(h_y_half[i]);

    // Compare results and calculate error
    std::cout << "\n=== Result Comparison ===" << std::endl;
    float max_error = 0.0f;
    float avg_error = 0.0f;
    float max_relative_error = 0.0f;
    int error_count = 0;
    constexpr float ERROR_THRESHOLD = 1e-5f;

    for (int i = 0; i < N; ++i) {
        float error = fabsf(h_y_gpu[i] - h_y[i]);
        float relative_error = h_y[i] != 0.0f ? error / fabsf(h_y[i]) : error;

        max_error = fmaxf(max_error, error);
        max_relative_error = fmaxf(max_relative_error, relative_error);
        avg_error += error;

        if (error > ERROR_THRESHOLD) {
            error_count++;
            if (error_count <= 5) { // Show first 5 errors
                std::cout << "Error at index " << i << ": GPU=" << h_y_gpu[i]
                          << ", CPU=" << h_y[i] << ", diff=" << error << std::endl;
            }
        }
    }

    avg_error /= N;

    std::cout << "\nError Statistics:" << std::endl;
    std::cout << "  Max Error        : " << max_error << std::endl;
    std::cout << "  Average Error    : " << avg_error << std::endl;
    std::cout << "  Max Relative Err : " << max_relative_error * 100.0f << "%" << std::endl;
    std::cout << "  Error Count (>" << ERROR_THRESHOLD << ") : " << error_count << " / " << N << std::endl;

    if (max_error < 1e-4f) {
        std::cout << "\nTest PASSED! Results match within tolerance." << std::endl;
    } else {
        std::cout << "\nTest FAILED! Errors exceed tolerance." << std::endl;
    }

    // Performance summary
    std::cout << "\n=== Performance Summary ===" << std::endl;
    std::cout << "CPU Time  : " << cpu_time << " ms" << std::endl;
    std::cout << "GPU Time  : " << compute_time << " ms" << std::endl;
    std::cout << "Speedup   : " << cpu_time / compute_time << "x" << std::endl;

    // Verify sum of softmax output (each token should sum to ~1.0)
    std::cout << "\nSoftmax sum check (per token, should be ~1.0):" << std::endl;
    float max_sum_error = 0.0f;
    for (int token = 0; token < TOKENS; ++token) {
        int base_idx = token * TOKEN_DIM;
        float token_sum = 0.0f;
        for (int i = 0; i < TOKEN_DIM; ++i) {
            token_sum += h_y_gpu[base_idx + i];
        }
        float sum_error = fabsf(token_sum - 1.0f);
        max_sum_error = fmaxf(max_sum_error, sum_error);
        if (token < 3) { // Show first 3 tokens
            std::cout << "  Token " << token << " sum: " << token_sum << std::endl;
        }
    }
    std::cout << "  Max sum error: " << max_sum_error << std::endl;

    // Cleanup - free GPU memory
    std::cout << "\n=== Cleanup ===" << std::endl;
    hipFree(d_x);
    hipFree(d_y);
    hiprandDestroyGenerator(gen);

    std::cout << "GPU memory freed successfully." << std::endl;
    std::cout << "Test completed!" << std::endl;

    return 0;
}