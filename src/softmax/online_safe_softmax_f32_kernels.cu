#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <thread>
#include <hiprand.h>

#define WARP_SIZE 32
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])


// Scale uniform [0,1) to [min, max)
__global__ void scale_uniform_kernel(float *data, int N, float min_val, float max_val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data[idx] = data[idx] * (max_val - min_val) + min_val;
    }
}

struct __align__(8) MD {
    float m;
    float d;
};

// Warp Reduce for Online Softmax
template <const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ MD warp_reduce_md_op(MD value) {
    unsigned int mask = 0xffffffff;
#pragma unroll
    for (int stride = kWarpSize >> 1; stride > 0; stride >>= 1) {
        MD other{};
        other.m = __shfl_xor_sync(mask, value.m, stride);
        other.d = __shfl_xor_sync(mask, value.d, stride);
        bool is_big = value.m > other.m;
        MD bigger_md = is_big ? value : other;
        MD smaller_md = is_big ? other : value;
        value.m = bigger_md.m;
        value.d = bigger_md.d + smaller_md.d * __expf(smaller_md.m - bigger_md.m);
    }
    return value;
}

// reference: https://arxiv.org/pdf/1805.02867
// widely used in flash-attn
template<const int NUM_THREADS = 256>
__global__ void online_safe_softmax_f32_per_token_kernel(const float *x, float *y, int N) {
    const int tid = threadIdx.x;
    const int idx = blockIdx.x * NUM_THREADS + tid;
    constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
    __shared__ MD reduce_md[NUM_WARPS];
    int warp = tid / WARP_SIZE;
    int lane = tid % WARP_SIZE;
    MD val;
    val.m = idx < N ? x[idx] : -FLT_MAX;
    val.d = idx < N ? 1.0f : 0.0f; // e^(xi-xi) = e^0 = 1
    val = warp_reduce_md_op<WARP_SIZE>(val);
    if (lane == 0) reduce_md[warp] = val;
    __syncthreads();
    // calculate the block reduce MD
    MD block_res = lane < NUM_WARPS ? reduce_md[lane] : MD{-FLT_MAX, 0};
    if (warp == 0) block_res = warp_reduce_md_op<NUM_WARPS>(block_res);
    if (tid == 0) reduce_md[0] = block_res;
    __syncthreads();
    // calculate the softmax value
    MD final_res = reduce_md[0];
    float d_total_inverse = __fdividef(1.0, final_res.d);
    if (idx < N) y[idx] = __expf(x[idx] - final_res.m) * d_total_inverse;
}

template<const int NUM_THREADS = 256 / 4>
__global__ void online_safe_softmax_f32x4_per_token_kernel(float *x, float *y, int N) {
    const int tid = threadIdx.x;
    const int idx = (blockIdx.x * NUM_THREADS + tid) * 4;
    constexpr int NUM_WARPS = (NUM_THREADS + WARP_SIZE - 1) / WARP_SIZE;
    __shared__ MD reduce_md[NUM_WARPS];
    int warp = tid / WARP_SIZE;
    int lane = tid % WARP_SIZE;
    float4 reg_x = FLOAT4(x[idx]);
    float mx_val = fmaxf(fmaxf(reg_x.x, reg_x.y), fmaxf(reg_x.z, reg_x.w));
    float d_val = __expf(reg_x.x - mx_val) + __expf(reg_x.y - mx_val) + __expf(reg_x.z - mx_val) + __expf(reg_x.w - mx_val);
    MD md{mx_val, d_val};
    MD res = warp_reduce_md_op<WARP_SIZE>(md);
    if (lane == 0) reduce_md[warp] = res;
    __syncthreads();
    MD block_res = lane < NUM_WARPS ? reduce_md[lane] : MD{-FLT_MAX, 0};
    if (warp == 0) block_res = warp_reduce_md_op<NUM_WARPS>(block_res);
    if (tid == 0) reduce_md[0] = block_res;
    __syncthreads();
    MD final_res = reduce_md[0];
    float d_total_inverse = __fdividef(1.0, final_res.d);
    float4 reg_y;
    reg_y.x = __expf(reg_x.x - final_res.m) * d_total_inverse;
    reg_y.y = __expf(reg_x.y - final_res.m) * d_total_inverse;
    reg_y.z = __expf(reg_x.z - final_res.m) * d_total_inverse;
    reg_y.w = __expf(reg_x.w - final_res.m) * d_total_inverse;
    if (idx + 3 < N) FLOAT4(y[idx]) = reg_y;
}


int main() {
    std::cout << "Start Testing Softmax kernels" << std::endl;
    constexpr int TOKENS = 2048;      // Number of tokens
    constexpr int TOKEN_DIM = 256;    // Dimension per token
    constexpr int N = TOKENS * TOKEN_DIM; // Total elements

    std::vector<float> h_y(N);
    std::vector<float> h_x(N);

    auto start_gen = std::chrono::high_resolution_clock::now();

    float *d_x;
    float *d_y;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    // gen
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1233ULL);
    hiprandGenerateUniform(gen, d_x, N);

    // scale to [0, 1]
    constexpr float MIN_VAL = 0.0f;
    constexpr float MAX_VAL = 2;
    scale_uniform_kernel<<<TOKENS, TOKEN_DIM>>>(d_x, N, MIN_VAL, MAX_VAL);

    hipDeviceSynchronize();

    auto end_gen = std::chrono::high_resolution_clock::now();
    auto gen_time = std::chrono::duration<double, std::milli>(end_gen - start_gen).count();
    std::cout << "GPU Random Number Generate Time : " << gen_time << " ms" << std::endl;

    // copy to host
    auto start_copy = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_x.data(), d_x, N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    auto end_copy = std::chrono::high_resolution_clock::now();
    auto copy_time = std::chrono::duration<double, std::milli>(end_copy - start_copy).count();
    std::cout << "GPU Copy data Time : " << copy_time << " ms" << std::endl;

    std::cout << "Start Compute softmax" << std::endl;
    auto start_compute = std::chrono::high_resolution_clock::now();

    // kernel functions
    // online_safe_softmax_f32_per_token_kernel<<<TOKENS, TOKEN_DIM>>>(d_x, d_y, N); // 40.0868x speed up
    online_safe_softmax_f32x4_per_token_kernel<<<TOKENS, TOKEN_DIM / 4>>>(d_x, d_y, N); // 45.2075x speed up
    hipDeviceSynchronize();

    auto end_compute = std::chrono::high_resolution_clock::now();
    auto compute_time = std::chrono::duration<double, std::milli>(end_compute - start_compute).count();
    std::cout << "GPU Compute softmax Time : " << compute_time << " ms" << std::endl;

    // CPU reference computation
    std::cout << "Start CPU Reference Compute" << std::endl;
    auto start_cpu = std::chrono::high_resolution_clock::now();

    // Per-token softmax: each token independently normalized
    for (int token = 0; token < TOKENS; ++token) {
        int base_idx = token * TOKEN_DIM;

        // Calculate sum of exp for this token
        float exp_sum = 0.0f;
        for (int i = 0; i < TOKEN_DIM; ++i) {
            exp_sum += expf(h_x[base_idx + i]);
        }

        // Normalize for this token
        for (int i = 0; i < TOKEN_DIM; ++i) {
            h_y[base_idx + i] = expf(h_x[base_idx + i]) / exp_sum;
        }
    }

    auto end_cpu = std::chrono::high_resolution_clock::now();
    auto cpu_time = std::chrono::duration<double, std::milli>(end_cpu - start_cpu).count();
    std::cout << "CPU Reference Compute Time : " << cpu_time << " ms" << std::endl;

    // Copy GPU result back to host
    std::vector<float> h_y_gpu(N);
    hipMemcpy(h_y_gpu.data(), d_y, N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Compare results and calculate error
    std::cout << "\n=== Result Comparison ===" << std::endl;
    float max_error = 0.0f;
    float avg_error = 0.0f;
    float max_relative_error = 0.0f;
    int error_count = 0;
    constexpr float ERROR_THRESHOLD = 1e-5f;

    for (int i = 0; i < N; ++i) {
        float error = fabsf(h_y_gpu[i] - h_y[i]);
        float relative_error = h_y[i] != 0.0f ? error / fabsf(h_y[i]) : error;

        max_error = fmaxf(max_error, error);
        max_relative_error = fmaxf(max_relative_error, relative_error);
        avg_error += error;

        if (error > ERROR_THRESHOLD) {
            error_count++;
            if (error_count <= 5) { // Show first 5 errors
                std::cout << "Error at index " << i << ": GPU=" << h_y_gpu[i]
                          << ", CPU=" << h_y[i] << ", diff=" << error << std::endl;
            }
        }
    }

    avg_error /= N;

    std::cout << "\nError Statistics:" << std::endl;
    std::cout << "  Max Error        : " << max_error << std::endl;
    std::cout << "  Average Error    : " << avg_error << std::endl;
    std::cout << "  Max Relative Err : " << max_relative_error * 100.0f << "%" << std::endl;
    std::cout << "  Error Count (>" << ERROR_THRESHOLD << ") : " << error_count << " / " << N << std::endl;

    if (max_error < 1e-4f) {
        std::cout << "\nTest PASSED! Results match within tolerance." << std::endl;
    } else {
        std::cout << "\nTest FAILED! Errors exceed tolerance." << std::endl;
    }

    // Performance summary
    std::cout << "\n=== Performance Summary ===" << std::endl;
    std::cout << "CPU Time  : " << cpu_time << " ms" << std::endl;
    std::cout << "GPU Time  : " << compute_time << " ms" << std::endl;
    std::cout << "Speedup   : " << cpu_time / compute_time << "x" << std::endl;

    // Verify sum of softmax output (each token should sum to ~1.0)
    std::cout << "\nSoftmax sum check (per token, should be ~1.0):" << std::endl;
    float max_sum_error = 0.0f;
    for (int token = 0; token < TOKENS; ++token) {
        int base_idx = token * TOKEN_DIM;
        float token_sum = 0.0f;
        for (int i = 0; i < TOKEN_DIM; ++i) {
            token_sum += h_y_gpu[base_idx + i];
        }
        float sum_error = fabsf(token_sum - 1.0f);
        max_sum_error = fmaxf(max_sum_error, sum_error);
        if (token < 3) { // Show first 3 tokens
            std::cout << "  Token " << token << " sum: " << token_sum << std::endl;
        }
    }
    std::cout << "  Max sum error: " << max_sum_error << std::endl;

    // Cleanup - free GPU memory
    std::cout << "\n=== Cleanup ===" << std::endl;
    hipFree(d_x);
    hipFree(d_y);
    hiprandDestroyGenerator(gen);

    std::cout << "GPU memory freed successfully." << std::endl;
    std::cout << "Test completed!" << std::endl;

    return 0;
}