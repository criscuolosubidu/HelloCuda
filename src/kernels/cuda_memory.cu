#include <iostream>
#include <hip/hip_runtime.h>
#include "cuda_memory.h"


template <typename T>
CudaMemory<T>::CudaMemory(size_t count) : count_(count)
{
    if (count > 0)
    {
        hipError_t err = hipMalloc(&data_, sizeof(T) * count_);
        if (err != hipSuccess)
        {
            throw std::runtime_error("Failed to allocate CUDA memory.");
        }
    }
}


template <typename T>
CudaMemory<T>::~CudaMemory()
{
    if (data_)
    {
        hipFree(data_);
    }
}


template <typename T>
CudaMemory<T>::CudaMemory(CudaMemory&& other) noexcept : data_(other.data_), count_(other.count_)
{
    other.data_ = nullptr;
    other.count_ = 0;
}


template <typename T>
CudaMemory<T>& CudaMemory<T>::operator=(CudaMemory&& other) noexcept
{
    if (this != &other)
    {
        if (data_)
        {
            hipFree(data_);
        }
        data_ = other.data_;
        count_ = other.count_;
        other.data_ = nullptr;
        other.count_ = 0;
    }
    return *this;
}

template class CudaMemory<float>;
template class CudaMemory<double>;
template class CudaMemory<int>;