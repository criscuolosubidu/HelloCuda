#include <hip/hip_runtime.h>
#include <iostream>

static constexpr int NUM_THREADS = 1024;
static constexpr int NUM_BLOCKS = 1024;

__global__ void incrementCounterNonAtomic(int* counter)
{
    int old = *counter;
    *counter = old + 1;
}

__global__ void incrementCounterAtomic(int* counter)
{
    atomicAdd(counter, 1);
}

void compareIncrementCounter()
{
    int h_counterNonAtomic = 0;
    int h_counterAtomic = 0;
    int *d_counterNonAtomic, *d_counterAtmoic;

    hipMalloc(reinterpret_cast<void**>(&d_counterNonAtomic), sizeof(int));
    hipMalloc(reinterpret_cast<void**>(&d_counterAtmoic), sizeof(int));

    hipMemcpy(d_counterNonAtomic, &h_counterNonAtomic, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_counterAtmoic, &h_counterAtomic, sizeof(int), hipMemcpyHostToDevice);

    incrementCounterNonAtomic<<<NUM_BLOCKS, NUM_THREADS>>>(d_counterNonAtomic);
    incrementCounterAtomic<<<NUM_BLOCKS, NUM_THREADS>>>(d_counterAtmoic);

    hipMemcpy(&h_counterNonAtomic, d_counterNonAtomic, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_counterAtomic, d_counterAtmoic, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Non-atomic counter value : " << h_counterNonAtomic << std::endl;
    std::cout << "Atomic counter value : " << h_counterAtomic << std::endl;

    hipFree(d_counterNonAtomic);
    hipFree(d_counterAtmoic);
}

